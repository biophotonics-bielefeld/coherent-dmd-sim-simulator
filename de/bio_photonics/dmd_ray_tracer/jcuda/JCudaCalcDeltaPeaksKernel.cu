#include "hip/hip_runtime.h"
#include "cudaUtils.h"
#include <hip/hip_complex.h>

extern "C" __global__ void calcDeltaPeaks(int n, double m, double ax, double ay,
        float *finalField) {
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) {
        return;
    }
    int mx = nrX - 1;
    int my = nrY - 1;
    
    double out[3];
    calcSphericalOut(i, out);
    double bx = out[0];
    double by = out[1];
    
    double arg0;
    if (ax == bx) {
        arg0 = -((2*by*m*my*M_PI)/lambda);
    } else if (ay == by) {
        arg0 = -((2*bx*m*mx*M_PI)/lambda);
    } else if (ax == bx && ay == by) {
        finalField[2*i] = nrX*nrY;
        finalField[2*i+1] = 0;
        return;
    } else arg0 = -((2*m*(bx*mx+by*my)*M_PI)/lambda);
    
    double arg1 = (2*ax*m*(1+mx)*M_PI)/lambda;
    double arg2 = (2*bx*m*(1+mx)*M_PI)/lambda;
    double arg3 = (2*ay*m*(1+my)*M_PI)/lambda;
    double arg4 = (2*by*m*(1+my)*M_PI)/lambda;
    double arg5 = (2*ax*m*M_PI)/lambda;
    double arg6 = (2*bx*m*M_PI)/lambda;
    double arg7 = (2*ay*m*M_PI)/lambda;
    double arg8 = (2*by*m*M_PI)/lambda;
    
    hipDoubleComplex z0 = make_hipDoubleComplex(cos(arg0), sin(arg0));
    hipDoubleComplex z1;
    hipDoubleComplex z2;
    hipDoubleComplex z3;
    hipDoubleComplex z4;
    
    if (ax == bx) {
        z1 = make_hipDoubleComplex(nrX, 0);
        z2 = make_hipDoubleComplex(
                cos(arg3)-cos(arg4), sin(arg3)-sin(arg4));
        z3 = make_hipDoubleComplex(1, 0);
        z4 = make_hipDoubleComplex(
                cos(arg7)-cos(arg8), sin(arg7)-sin(arg8));
    } else if (ay == by) {
        z1 = make_hipDoubleComplex(
                cos(arg1)-cos(arg2), sin(arg1)-sin(arg2));
        z2 = make_hipDoubleComplex(nrY, 0);
        z3 = make_hipDoubleComplex(
                cos(arg5)-cos(arg6), sin(arg5)-sin(arg6));
        z4 = make_hipDoubleComplex(1, 0);
    } else {
        z1 = make_hipDoubleComplex(
                cos(arg1)-cos(arg2), sin(arg1)-sin(arg2));
        z2 = make_hipDoubleComplex(
                cos(arg3)-cos(arg4), sin(arg3)-sin(arg4));
        z3 = make_hipDoubleComplex(
                cos(arg5)-cos(arg6), sin(arg5)-sin(arg6));
        z4 = make_hipDoubleComplex(
                cos(arg7)-cos(arg8), sin(arg7)-sin(arg8));
    }
    
    z0 = hipCmul(z0, z1);
    z0 = hipCmul(z0, z2);
    z3 = hipCmul(z3, z4);
    z0 = hipCdiv(z0, z3);
    
    finalField[2*i] = static_cast<float>(z0.x);
    finalField[2*i + 1] = static_cast<float>(z0.y);
}