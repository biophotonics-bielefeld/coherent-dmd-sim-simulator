#include "hip/hip_runtime.h"
#include "cudaUtils.h"

/** cuda kernel for the dmd ray tracer
 * calculates the field for all out angles
 * @param n number of out angles/pixels
 * @param mStart index of first mirror for this kernel
 * @param mEnd index of (last mirror + 1) for this kernel
 * @param *tiltStates pointer to the array of tiltStates (1&0 as true/false)
 * @param *mirrorTrue pointer to the array of the reference field for true mirrors
 * @param *mirrorFalse pointer to the array of the reference field for false mirrors
 * @param *inOffsetPathLengths pointer to the array which contains the pathlength
 * between the incoming wave and each mirror
 * @param *beamProfile pointer to the array of in beam intensitys for each mirror
 * @param *finalField pointer to the array for the resulting field for each out angle
 */

extern "C" __global__ void calcOutAngles(int n, int mStart, int mEnd, int *tiltStates,
		float *mirrorTrue, float *mirrorFalse, float *inOffsetPathLengths, float *beamProfile, float *finalField) {
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) {
        return;
    }
    
    float out[3];
    calcSphericalOut(i, out);
    //calcCartesianOut(i, out);
    
    for (int m = mStart; m < mEnd; m++) {
        float* referenceMirror = tiltStates[m] ? mirrorTrue : mirrorFalse;
        float referenceFieldRe = referenceMirror[2 * i];
        float referenceFieldIm = referenceMirror[2 * i + 1];
        float referenceFieldAbs = calcComplexAbs(referenceFieldRe, referenceFieldIm);
        float referenceFieldArg = atan2(referenceFieldIm, referenceFieldRe);
        
        int my = m / nrX;
        int mx = m % nrX;
        float dmdPosition[3];
        calcDmdPosition(mx, my, dmdPosition);
        
        float initInPathLength = inOffsetPathLengths[m];
        float outPathLength = calcOutPathLength(out, dmdPosition);
        float additionalPl = initInPathLength + outPathLength;
        float additionalPhase = (additionalPl / lambda) * 2 * M_PI;
        float r = referenceFieldAbs * beamProfile[m];
        float p = referenceFieldArg + additionalPhase;
        float finalFieldRe = r * cos(p);
        float finalFieldIm = r * sin(p);
        finalField[2*i] += finalFieldRe;
        finalField[2*i + 1] += finalFieldIm;
    }
}