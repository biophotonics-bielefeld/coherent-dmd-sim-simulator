#include "hip/hip_runtime.h"
/*
 * This file is part of the coherent_dmd_cimulator.
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */
#include "cudaUtils.h"
#include <hip/hip_complex.h>

extern "C" __global__ void calcSingleMirror(int n, double m,
        double ax, double ay, double az, double alpha, float *finalField) {
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;
    
    double out[3];
    calcSphericalOut(i, out);
    double bx = out[0];
    double by = out[1];
    double bz = out[2];
    
    double s2 = sqrtf(2);
    double ca = __cosf(alpha);
    double sa = __sinf(alpha);

    double r0 = lambda*lambda;
    double r1 = M_PI*M_PI;
    double r2 = ax+ay-bx-by+(ax-ay-bx+by)*ca-s2*(az-bz)*sa;
    double r3 = -ax-ay+bx+by+(ax-ay-bx+by)*ca-s2*(az-bz)*sa;
    double r = r0/r1/r2/r3;

    //double arg0 = 0;
    double arg1 = (2*ax*m+2*ay*m-2*bx*m-2*by*m)*M_PI/lambda;
    double arg2 = (ax*m+ay*m-bx*m-by*m+(ax-ay-bx+by)*m*ca-s2*(az-bz)*m*sa)*M_PI/lambda;
    double arg3 = (ax*m+ay*m-bx*m-by*m-(ax-ay-bx+by)*m*ca+s2*(az-bz)*m*sa)*M_PI/lambda;

    double re0 = 1;
    double im0 = 0;
    double re1 = __cosf(arg1);
    double im1 = __sinf(arg1);
    double re2 = __cosf(arg2);
    double im2 = __sinf(arg2);
    double re3 = __cosf(arg3);
    double im3 = __sinf(arg3);

    double nx = 1/s2*sa;
    double ny = -nx;
    double nz = sqrtf(1-sa*sa);
    double intesityFactor = fabsf(ax*nx+ay*ny+az*nz);

    finalField[2*i] = static_cast<float>(intesityFactor * r * (re0 + re1 - re2 - re3));
    finalField[2*i + 1] = static_cast<float>(intesityFactor * r * (im0 + im1 - im2 - im3));
    //if (i == 994119) printf("GPU: %f %f %f %f %f \n", -ax-ay+bx+by, (ax-ay-bx+by)*ca, s2*(az-bz)*sa, r3, r);
}