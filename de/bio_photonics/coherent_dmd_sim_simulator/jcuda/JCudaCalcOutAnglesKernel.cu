#include "hip/hip_runtime.h"
/*
 * This file is part of the coherent_dmd_cimulator.
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */
#include "cudaUtils.h"

/** cuda kernel for the dmd ray tracer
 * calculates the field for all out angles
 * @param n number of out angles/pixels
 * @param mStart index of first mirror for this kernel
 * @param mEnd index of (last mirror + 1) for this kernel
 * @param *tiltStates pointer to the array of tiltStates (1&0 as true/false)
 * @param *mirrorTrue pointer to the array of the reference field for true mirrors
 * @param *mirrorFalse pointer to the array of the reference field for false mirrors
 * @param *inOffsetPathLengths pointer to the array which contains the pathlength
 * between the incoming wave and each mirror
 * @param *beamProfile pointer to the array of in beam intensitys for each mirror
 * @param *finalField pointer to the array for the resulting field for each out angle
 */

extern "C" __global__ void calcOutAngles(int n, int mStart, int mEnd, int *tiltStates,
		float *mirrorTrue, float *mirrorFalse, float *inOffsetPathLengths, float *beamProfile, float *finalField) {
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) {
        return;
    }
    
    float out[3];
    calcSphericalOut(i, out);
    //calcCartesianOut(i, out);
    
    for (int m = mStart; m < mEnd; m++) {
        float* referenceMirror = tiltStates[m] ? mirrorTrue : mirrorFalse;
        float referenceFieldRe = referenceMirror[2 * i];
        float referenceFieldIm = referenceMirror[2 * i + 1];
        float referenceFieldAbs = calcComplexAbs(referenceFieldRe, referenceFieldIm);
        float referenceFieldArg = atan2(referenceFieldIm, referenceFieldRe);
        
        int my = m / nrX;
        int mx = m % nrX;
        float dmdPosition[3];
        calcDmdPosition(mx, my, dmdPosition);
        
        float initInPathLength = inOffsetPathLengths[m];
        float outPathLength = calcOutPathLength(out, dmdPosition);
        float additionalPl = initInPathLength + outPathLength;
        float additionalPhase = (additionalPl / lambda) * 2 * M_PI;
        float r = referenceFieldAbs * beamProfile[m];
        float p = referenceFieldArg + additionalPhase;
        float finalFieldRe = r * cos(p);
        float finalFieldIm = r * sin(p);
        finalField[2*i] += finalFieldRe;
        finalField[2*i + 1] += finalFieldIm;
    }
}